#include "include_arma.h"
#include <hipsolver.h>

int solve_positive_definite_gpu(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx)
{
	hipDeviceReset();

	// device sparse matrix A to device (CSC format)
	int * dA_col_ptrs, * dA_row_indices;
	real_t * dA_values;
	
	hipMalloc(&dA_col_ptrs, (m + 1) * sizeof(int));
	hipMemcpy(dA_col_ptrs, hA_col_ptrs, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&dA_row_indices, nnz * sizeof(int));
	hipMemcpy(dA_row_indices, hA_row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&dA_values, nnz * sizeof(real_t));
	hipMemcpy(dA_values, hA_values, nnz * sizeof(real_t), hipMemcpyHostToDevice); 
	
	// vector b to device
	real_t * db;
	hipMalloc(&db, nnz * sizeof(real_t));
	hipMemcpy(db, hb, nnz * sizeof(real_t), hipMemcpyHostToDevice);

	// vector x to device
	real_t * dx;
	hipMalloc(&dx, m * sizeof(real_t));

	// solve Ax = b

	int singularity;
	
	hipsolverSpHandle_t handle_cusolver;
	hipsolverSpCreate(&handle_cusolver);

	hipsparseMatDescr_t descr = 0;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	
#ifdef SINGLE_P
	hipsolverSpScsrlsvchol(handle_cusolver, m, nnz, descr, dA_values, dA_col_ptrs, dA_row_indices, db, 0, 0, dx, &singularity);
#else
	hipsolverSpDcsrlsvchol(handle_cusolver, m, nnz, descr, dA_values, dA_col_ptrs, dA_row_indices, db, 0, 0, dx, &singularity);
#endif
	
	hipsparseDestroyMatDescr(descr);
	hipsolverSpDestroy(handle_cusolver);
	
	// copy dx to host x
	hipMemcpy(hx, dx, m * sizeof(real_t), hipMemcpyDeviceToHost);

	// free device memory
	hipFree(dA_col_ptrs);
	hipFree(dA_row_indices);
	hipFree(dA_values);
	hipFree(db);
	hipFree(dx);
	
	return singularity;
}

