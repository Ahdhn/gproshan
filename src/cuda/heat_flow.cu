#include "include_arma.h"
#include <hipsolver.h>

int solve_positive_definite_gpu(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx)
{
	hipDeviceReset();

	// device sparse matrix A to device (CSC format)
	int * dA_col_ptrs, * dA_row_indices;
	real_t * dA_values;
	
	hipMalloc(&dA_col_ptrs, (m + 1) * sizeof(int));
	hipMemcpy(dA_col_ptrs, hA_col_ptrs, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&dA_row_indices, nnz * sizeof(int));
	hipMemcpy(dA_row_indices, hA_row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&dA_values, nnz * sizeof(real_t));
	hipMemcpy(dA_values, hA_values, nnz * sizeof(real_t), hipMemcpyHostToDevice); 
	
	// vector b to device
	real_t * db;
	hipMalloc(&db, nnz * sizeof(real_t));
	hipMemcpy(db, hb, nnz * sizeof(real_t), hipMemcpyHostToDevice);

	// vector x to device
	real_t * dx;
	hipMalloc(&dx, m * sizeof(real_t));

	// solve Ax = b with Cholesky factorization

	int singularity;
	
	hipsparseHandle_t handle;
	hipsparseCreate(&handle);

	hipsparseMatDescr_t descr = 0;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	csric02Info_t info;
	hipsparseCreateCsric02Info(&info);
	
	int buffer_size;
	void * buffer;

#ifdef SINGLE_P
#else
	hipsparseDcsric02_bufferSize(handle, m, nnz, descr, dA_values, dA_col_ptrs, dA_row_indices, info, &buffer_size);

	hipMalloc(&buffer, buffer_size);
	hipsparseDcsric02_analysis(handle, m, nnz, descr, dA_values, dA_col_ptrs, dA_row_indices, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);

	hipsparseDcsric02(handle, m, nnz, descr, dA_values, dA_col_ptrs, dA_row_indices, info, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);


#endif
	
	// copy dx to host x
	hipMemcpy(hx, dx, m * sizeof(real_t), hipMemcpyDeviceToHost);
	
	// destroy
	hipsparseDestroyCsric02Info(info);
	hipsparseDestroyMatDescr(descr);
	hipsparseDestroy(handle);

	// free device memory
	hipFree(buffer);
	hipFree(dA_col_ptrs);
	hipFree(dA_row_indices);
	hipFree(dA_values);
	hipFree(db);
	hipFree(dx);
	
	return singularity;
}

