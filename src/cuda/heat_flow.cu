#include "hip/hip_runtime.h"
#include "include_arma.h"

#include <cassert>

#include <omp.h>
#include <hipsolver.h>

double solve_positive_definite_gpu(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx)
{
	hipDeviceReset();

	// device sparse matrix A to device (CSC format)
	int * dA_col_ptrs, * dA_row_indices;
	real_t * dA_values;
	
	hipMalloc(&dA_col_ptrs, (m + 1) * sizeof(int));
	hipMemcpy(dA_col_ptrs, hA_col_ptrs, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&dA_row_indices, nnz * sizeof(int));
	hipMemcpy(dA_row_indices, hA_row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&dA_values, nnz * sizeof(real_t));
	hipMemcpy(dA_values, hA_values, nnz * sizeof(real_t), hipMemcpyHostToDevice); 
	
	// vector b to device
	real_t * db;
	hipMalloc(&db, nnz * sizeof(real_t));
	hipMemcpy(db, hb, nnz * sizeof(real_t), hipMemcpyHostToDevice);

	// vector x to device
	real_t * dx;
	hipMalloc(&dx, m * sizeof(real_t));
	
	// aux vector y to device
	real_t * dy;
	hipMalloc(&dy, m * sizeof(real_t));
	
	hipsparseHandle_t handle;
	hipsparseCreate(&handle);

	// SOLVE Ax = b
	double solve_time;

	hipsparseMatDescr_t descr_M = 0;
	hipsparseMatDescr_t descr_L = 0;
	
	csric02Info_t info_M = 0;
	csrsv2Info_t info_L = 0;
	csrsv2Info_t info_Lt = 0;
	
	int buffer_size_M;
	int buffer_size_L;
	int buffer_size_Lt;
	int buffer_size;
	
	void * buffer = 0;

	int structural_zero;
	int numerical_zero;

	const real_t alpha = 1.;
	const hipsparseSolvePolicy_t policy_M  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
	const hipsparseSolvePolicy_t policy_L  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
	const hipsparseSolvePolicy_t policy_Lt = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
	const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	const hipsparseOperation_t trans_Lt = HIPSPARSE_OPERATION_TRANSPOSE;

	hipsparseCreateMatDescr(&descr_M);
	hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

	hipsparseCreateMatDescr(&descr_L);
	hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);

	hipsparseCreateCsric02Info(&info_M);
	hipsparseCreateCsrsv2Info(&info_L);
	hipsparseCreateCsrsv2Info(&info_Lt);

#ifdef SINGLE_P
#else
	hipsparseDcsric02_bufferSize(handle, m, nnz, descr_M, dA_values, dA_col_ptrs, dA_row_indices, info_M, &buffer_size_M);
	hipsparseDcsrsv2_bufferSize(handle, trans_L, m, nnz, descr_L, dA_values, dA_col_ptrs, dA_row_indices, info_L, &buffer_size_L);
	hipsparseDcsrsv2_bufferSize(handle, trans_Lt, m, nnz, descr_L, dA_values, dA_col_ptrs, dA_row_indices, info_Lt, &buffer_size_Lt);
#endif

	buffer_size = max(buffer_size_M, max(buffer_size_L, buffer_size_Lt));
	printf("%d\n", buffer_size);
	hipMalloc(&buffer, buffer_size);

#ifdef SINGLE_P
#else
	hipsparseDcsric02_analysis(handle, m, nnz, descr_M, dA_values, dA_col_ptrs, dA_row_indices, info_M, policy_M, buffer);
#endif
	if(HIPSPARSE_STATUS_ZERO_PIVOT == hipsparseXcsric02_zeroPivot(handle, info_M, &structural_zero))
		printf("A(%d,%d) is missing\n", structural_zero, structural_zero);

#ifdef SINGLE_P
#else
	hipsparseDcsrsv2_analysis(handle, trans_L, m, nnz, descr_L, dA_values, dA_col_ptrs, dA_row_indices, info_L, policy_L, buffer);
	hipsparseDcsrsv2_analysis(handle, trans_Lt, m, nnz, descr_L, dA_values, dA_col_ptrs, dA_row_indices, info_Lt, policy_Lt, buffer);

	hipsparseDcsric02(handle, m, nnz, descr_M, dA_values, dA_col_ptrs, dA_row_indices, info_M, policy_M, buffer);
#endif
	if(HIPSPARSE_STATUS_ZERO_PIVOT == hipsparseXcsric02_zeroPivot(handle, info_M, &numerical_zero))
		printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);

	TIC(solve_time)

#ifdef SINGLE_P
#else
	assert(HIPSPARSE_STATUS_SUCCESS == hipsparseDcsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L, dA_values, dA_col_ptrs, dA_row_indices, info_L, db, dy, policy_L, buffer));
	assert(HIPSPARSE_STATUS_SUCCESS == hipsparseDcsrsv2_solve(handle, trans_Lt, m, nnz, &alpha, descr_L, dA_values, dA_col_ptrs, dA_row_indices, info_Lt, dy, dx, policy_Lt, buffer));
#endif
	
	// copy sol x to host
	hipMemcpy(hx, dx, m * sizeof(real_t), hipMemcpyDeviceToHost);

	TOC(solve_time)

	// FREE
	hipFree(buffer);
	hipsparseDestroyMatDescr(descr_M);
	hipsparseDestroyMatDescr(descr_L);
	hipsparseDestroyCsric02Info(info_M);
	hipsparseDestroyCsrsv2Info(info_L);
	hipsparseDestroyCsrsv2Info(info_Lt);
	hipsparseDestroy(handle);

	hipFree(dA_col_ptrs);
	hipFree(dA_row_indices);
	hipFree(dA_values);
	hipFree(db);
	hipFree(dx);
	hipFree(dx);
	
	return solve_time;
}

