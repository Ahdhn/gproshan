#include "hip/hip_runtime.h"
#include "include_arma.h"

#include <cassert>

#include <hipsolver.h>
#include <hipsolver.h>

struct cu_spAxb
{
	int * A_col_ptrs, * A_row_indices;
	real_t * A_values, * x, * b;
	
	cu_spAxb(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx)
	{
		hipMalloc(&A_col_ptrs, (m + 1) * sizeof(int));
		hipMemcpy(A_col_ptrs, hA_col_ptrs, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

		hipMalloc(&A_row_indices, nnz * sizeof(int));
		hipMemcpy(A_row_indices, hA_row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);

		hipMalloc(&A_values, nnz * sizeof(real_t));
		hipMemcpy(A_values, hA_values, nnz * sizeof(real_t), hipMemcpyHostToDevice); 
		
		hipMalloc(&b, nnz * sizeof(real_t));
		hipMemcpy(b, hb, nnz * sizeof(real_t), hipMemcpyHostToDevice);

		hipMalloc(&x, m * sizeof(real_t));
	}

	~cu_spAxb()
	{
		hipFree(A_col_ptrs);
		hipFree(A_row_indices);
		hipFree(A_values);
		hipFree(b);
		hipFree(x);
	}
};

double solve_positive_definite_cusolver(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx, const bool host)
{
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	// solve Ax = b

	int singularity;
	
	hipsolverSpHandle_t handle_cusolver;
	hipsolverSpCreate(&handle_cusolver);

	hipsparseMatDescr_t descr = 0;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	
	if(host)
	{
		#ifdef SINGLE_P
			hipsolverSpScsrlsvcholHost(handle_cusolver, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, hb, 0, 0, hx, &singularity);
		#else
			hipsolverSpDcsrlsvcholHost(handle_cusolver, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, hb, 0, 0, hx, &singularity);
		#endif
	}
	else
	{
		// allocate A, x, b into device
		cu_spAxb data(m, nnz, hA_values, hA_col_ptrs, hA_row_indices, hb, hx);

		#ifdef SINGLE_P
			hipsolverSpScsrlsvchol(handle_cusolver, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, data.b, 0, 0, data.x, &singularity);
		#else
			hipsolverSpDcsrlsvchol(handle_cusolver, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, data.b, 0, 0, data.x, &singularity);
		#endif
	
		hipMemcpy(hx, data.x, m * sizeof(real_t), hipMemcpyDeviceToHost);
	}

	printf("%d\n", singularity != -1);

	hipsparseDestroyMatDescr(descr);
	hipsolverSpDestroy(handle_cusolver);

	// end Ax = b

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (double) time / 1000;
}

double solve_positive_definite_cusparse(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx)
{
	hipDeviceReset();
	
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate A, x, b into device
	cu_spAxb data(m, nnz, hA_values, hA_col_ptrs, hA_row_indices, hb, hx);
	
	// aux vector y to device
	real_t * dy;
	hipMalloc(&dy, m * sizeof(real_t));
	
	hipsparseHandle_t handle;
	hipsparseCreate(&handle);

	// SOLVE Ax = b
	
	hipsparseMatDescr_t descr_M = 0;
	hipsparseMatDescr_t descr_L = 0;
	
	csric02Info_t info_M = 0;
	csrsv2Info_t info_L = 0;
	csrsv2Info_t info_Lt = 0;
	
	int buffer_size_M;
	int buffer_size_L;
	int buffer_size_Lt;
	int buffer_size;
	
	void * buffer = 0;

	int structural_zero;
	int numerical_zero;

	const real_t alpha = 1.;
	const hipsparseSolvePolicy_t policy_M  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
	const hipsparseSolvePolicy_t policy_L  = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
	const hipsparseSolvePolicy_t policy_Lt = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
	const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	const hipsparseOperation_t trans_Lt = HIPSPARSE_OPERATION_TRANSPOSE;

	hipsparseCreateMatDescr(&descr_M);
	hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

	hipsparseCreateMatDescr(&descr_L);
	hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);

	hipsparseCreateCsric02Info(&info_M);
	hipsparseCreateCsrsv2Info(&info_L);
	hipsparseCreateCsrsv2Info(&info_Lt);

	#ifdef SINGLE_P
		hipsparseScsric02_bufferSize(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, &buffer_size_M);
		hipsparseScsrsv2_bufferSize(handle, trans_L, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, &buffer_size_L);
		hipsparseScsrsv2_bufferSize(handle, trans_Lt, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, &buffer_size_Lt);
	#else
		hipsparseDcsric02_bufferSize(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, &buffer_size_M);
		hipsparseDcsrsv2_bufferSize(handle, trans_L, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, &buffer_size_L);
		hipsparseDcsrsv2_bufferSize(handle, trans_Lt, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, &buffer_size_Lt);
	#endif

	buffer_size = max(buffer_size_M, max(buffer_size_L, buffer_size_Lt));
	hipMalloc(&buffer, buffer_size);

	#ifdef SINGLE_P
		hipsparseScsric02_analysis(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, policy_M, buffer);
	#else
		hipsparseDcsric02_analysis(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, policy_M, buffer);
	#endif
	if(HIPSPARSE_STATUS_ZERO_PIVOT == hipsparseXcsric02_zeroPivot(handle, info_M, &structural_zero))
		printf("A(%d,%d) is missing\n", structural_zero, structural_zero);

	#ifdef SINGLE_P
		hipsparseScsrsv2_analysis(handle, trans_L, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, policy_L, buffer);
		hipsparseScsrsv2_analysis(handle, trans_Lt, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, policy_Lt, buffer);

		hipsparseScsric02(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, policy_M, buffer);
	#else
		hipsparseDcsrsv2_analysis(handle, trans_L, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, policy_L, buffer);
		hipsparseDcsrsv2_analysis(handle, trans_Lt, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, policy_Lt, buffer);

		hipsparseDcsric02(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, policy_M, buffer);
	#endif
	if(HIPSPARSE_STATUS_ZERO_PIVOT == hipsparseXcsric02_zeroPivot(handle, info_M, &numerical_zero))
		printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);


	// SOLVE
	hipEventRecord(start, 0);
	
	#ifdef SINGLE_P
		hipsparseScsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, data.b, dy, policy_L, buffer);
		hipsparseScsrsv2_solve(handle, trans_Lt, m, nnz, &alpha, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, dy, data.x, policy_Lt, buffer);
	#else
		hipsparseDcsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, data.b, dy, policy_L, buffer);
		hipsparseDcsrsv2_solve(handle, trans_Lt, m, nnz, &alpha, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, dy, data.x, policy_Lt, buffer);
	#endif
	
	// copy sol x to host
	hipMemcpy(hx, data.x, m * sizeof(real_t), hipMemcpyDeviceToHost);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// END SOLVE
	
	// FREE
	hipFree(buffer);
	hipsparseDestroyMatDescr(descr_M);
	hipsparseDestroyMatDescr(descr_L);
	hipsparseDestroyCsric02Info(info_M);
	hipsparseDestroyCsrsv2Info(info_L);
	hipsparseDestroyCsrsv2Info(info_Lt);
	hipsparseDestroy(handle);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (double) time / 1000;
}

double solve_positive_definite_gpu(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx)
{
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// device sparse matrix A to device (CSC format)
	/*
	int * dA_col_ptrs, * dA_row_indices;
	real_t * dA_values;
	
	hipMalloc(&dA_col_ptrs, (m + 1) * sizeof(int));
	hipMemcpy(dA_col_ptrs, hA_col_ptrs, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&dA_row_indices, nnz * sizeof(int));
	hipMemcpy(dA_row_indices, hA_row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&dA_values, nnz * sizeof(real_t));
	hipMemcpy(dA_values, hA_values, nnz * sizeof(real_t), hipMemcpyHostToDevice); 
	
	// vector b to device
	real_t * db;
	hipMalloc(&db, nnz * sizeof(real_t));
	hipMemcpy(db, hb, nnz * sizeof(real_t), hipMemcpyHostToDevice);

	// vector x to device
	real_t * dx;
	hipMalloc(&dx, m * sizeof(real_t));
	*/

	double solve_time;
	// SOLVE Ax = b

	hipsolverSpHandle_t cusolver_handle = NULL;
	hipsparseHandle_t cusparse_handle = NULL;
	hipStream_t stream = NULL;

	hipsparseMatDescr_t descr = NULL;

	csrcholInfoHost_t info;

	size_t size_iternal = 0;
	size_t size_chol = 0;

	void * buffer = NULL;

	int singularity;

	hipsolverSpCreate(&cusolver_handle);
	hipsparseCreate(&cusparse_handle);

	hipStreamCreate(&stream);
	hipsolverSpSetStream(cusolver_handle, stream);
	hipsparseSetStream(cusparse_handle, stream);
	
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	cusolverSpCreateCsrcholInfoHost(&info);

	cusolverSpXcsrcholAnalysisHost(cusolver_handle, m, nnz, descr, hA_col_ptrs, hA_row_indices, info);
	cusolverSpDcsrcholBufferInfoHost(cusolver_handle, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, info, &size_iternal, &size_chol);
	
	buffer = new char[size_chol];

	cusolverSpDcsrcholFactorHost(cusolver_handle, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, info, buffer);

	cusolverSpDcsrcholZeroPivotHost(cusolver_handle, info, 0, &singularity);
	assert(singularity == -1);

	// solve
	hipEventRecord(start, 0);
	
	cusolverSpDcsrcholSolveHost(cusolver_handle, m, hb, hx, info, buffer);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	solve_time = time / 1000;


	//hipMemcpy(hx, dx, m * sizeof(real_t), hipMemcpyDeviceToHost);

	// FREE
	delete [] (char*) buffer;
	cusolverSpDestroyCsrcholInfoHost(info);
	hipStreamDestroy(stream);
	hipsparseDestroyMatDescr(descr);
	hipsparseDestroy(cusparse_handle);
	hipsolverSpDestroy(cusolver_handle);
/*
	hipFree(dA_col_ptrs);
	hipFree(dA_row_indices);
	hipFree(dA_values);
	hipFree(db);
	hipFree(dx);
*/	
	return solve_time;
}

