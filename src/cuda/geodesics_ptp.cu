#include "hip/hip_runtime.h"
#include "geodesics_ptp.h"
#include "geodesics_ptp.cuh"
#include "che.cuh"

#include <cstdio>
#include <fstream>
#include <cassert>
#include <hipblas.h>

#define NT 32
#define NB(x) (x + NT - 1) / NT

distance_t * parallel_toplesets_propagation_gpu(che * mesh, const vector<index_t> & sources, const vector<index_t> & limits, const index_t * sorted_index, float & time_ptp, index_t * clusters)
{
	debug_me(GEODESICS_PTP)

	hipDeviceReset();

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// BEGIN PTP
	
	CHE * h_mesh = new CHE(mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	distance_t * h_dist = new distance_t[h_mesh->n_vertices];
	
	distance_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(distance_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(distance_t) * h_mesh->n_vertices);

	index_t * d_sorted;
	hipMalloc(&d_sorted, sizeof(index_t) * h_mesh->n_vertices);
	
	index_t d;
	if(clusters)
	{
		index_t * d_clusters[2] = {NULL, NULL};
		hipMalloc(&d_clusters[0], sizeof(index_t) * h_mesh->n_vertices);	
		hipMalloc(&d_clusters[1], sizeof(index_t) * h_mesh->n_vertices);	
	
		d = run_ptp_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, sources, limits, sorted_index, d_sorted, clusters, d_clusters);
		hipMemcpy(clusters, d_clusters[d], sizeof(index_t) * h_mesh->n_vertices, hipMemcpyDeviceToHost);

		hipFree(d_clusters[0]);
		hipFree(d_clusters[1]);
	}
	else
	{
		d = run_ptp_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, sources, limits, sorted_index, d_sorted);
	}

	hipMemcpy(h_dist, d_dist[d], sizeof(distance_t) * h_mesh->n_vertices, hipMemcpyDeviceToHost);
	
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	hipFree(d_sorted);
	cuda_free_CHE(dd_mesh, d_mesh);
	
	// END PTP

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_ptp, start, stop);
	time_ptp /= 1000;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return h_dist;
}

distance_t farthest_point_sampling_ptp_gpu(che * mesh, vector<index_t> & samples, float & time_fps, size_t n, distance_t radio)
{
	debug_me(GEODESICS_PTP)
	
	hipDeviceReset();

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// BEGIN FPS PTP

	CHE * h_mesh = new CHE(mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	distance_t * h_dist = new distance_t[h_mesh->n_vertices];
	
	distance_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(distance_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(distance_t) * h_mesh->n_vertices);

	index_t * d_sorted;
	hipMalloc(&d_sorted, sizeof(index_t) * h_mesh->n_vertices);
	
	vector<index_t> limits;
	index_t * toplesets = new index_t[h_mesh->n_vertices];
	index_t * sorted_index = new index_t[h_mesh->n_vertices];
	
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	if(n >= h_mesh->n_vertices) n = h_mesh->n_vertices >> 1;

	n -= samples.size();
	samples.reserve(n);
	
	index_t d;
	int f;
	distance_t max_dist = INFINITY;
	while(n-- && max_dist > radio)
	{
		limits.clear();
		mesh->sort_by_rings(toplesets, sorted_index, limits, samples);
		
		d = run_ptp_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, samples, limits, sorted_index, d_sorted);
				
		// 1 indexing
		#ifdef SINGLE_P
			hipblasIsamax(handle, mesh->n_vertices(), d_dist[d], 1, &f);
		#else
			hipblasIdamax(handle, mesh->n_vertices(), d_dist[d], 1, &f);
		#endif
		
		if(radio > 0 || !n)
			hipMemcpy(&max_dist, d_dist[d] + f - 1, sizeof(distance_t), hipMemcpyDeviceToHost);

		samples.push_back(f - 1);
	}
	
	hipblasDestroy(handle);
	
	delete [] h_dist;
	delete [] toplesets;
	delete [] sorted_index;

	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	hipFree(d_sorted);
	cuda_free_CHE(dd_mesh, d_mesh);
	
	// END FPS PTP

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_fps, start, stop);
	time_fps /= 1000;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return max_dist;
}

index_t run_ptp_gpu(CHE * d_mesh, const index_t & n_vertices, distance_t * h_dist, distance_t ** d_dist, const vector<index_t> & sources, const vector<index_t> & limits, const index_t * h_sorted, index_t * d_sorted, index_t * h_clusters, index_t ** d_clusters)
{
	#pragma omp parallel for
	for(index_t v = 0; v < n_vertices; v++)
		h_dist[v] = INFINITY;

	for(index_t i = 0; i < sources.size(); i++)
		h_dist[sources[i]] = 0;
		
	hipMemcpy(d_dist[0], h_dist, sizeof(distance_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_dist[1], h_dist, sizeof(distance_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_sorted, h_sorted, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);
	
	if(h_clusters)
	{
		assert(d_clusters);

		for(index_t i = 0; i < sources.size(); i++)
			h_clusters[sources[i]] = i + 1;

		hipMemcpy(d_clusters[0], h_clusters, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);
		hipMemcpy(d_clusters[1], h_clusters, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);
	}

	index_t d = 0;
	index_t start, end;
	index_t iter = iterations(limits);
	for(index_t i = 2; i < iter; i++)
	{
		start = start_v(i, limits);
		end = end_v(i, limits);
		
		if(h_clusters)
		{
			relax_ptp <<< NB(end - start), NT >>> (d_mesh, d_dist[!d], d_dist[d], d_clusters[!d], d_clusters[d], d_sorted, end, start);
		}
		else
			relax_ptp <<< NB(end - start), NT >>> (d_mesh, d_dist[!d], d_dist[d], d_sorted, end, start);
		hipDeviceSynchronize();
		d = !d;
	}
	
	return d;
}

__global__
void relax_ptp(CHE * mesh, distance_t * new_dist, distance_t * old_dist, index_t * sorted, index_t end, index_t start)
{
	index_t v = blockDim.x * blockIdx.x + threadIdx.x + start;
	
	if(v < end)
	{
		v = sorted ? sorted[v] : v;
		if(v < mesh->n_vertices)
		{
			new_dist[v] = old_dist[v];

			distance_t d;
			cu_for_star(he, mesh, v)
			{
				d = cu_update_step(mesh, old_dist, he);
				if(d < new_dist[v]) new_dist[v] = d;
			}
		}
	}
}


__global__
void relax_ptp(CHE * mesh, distance_t * new_dist, distance_t * old_dist, index_t * new_clusters, index_t * old_clusters, index_t * sorted, index_t end, index_t start)
{
	index_t v = blockDim.x * blockIdx.x + threadIdx.x + start;
	
	if(v < end)
	{
		v = sorted ? sorted[v] : v;
		if(v < mesh->n_vertices)
		{
			new_dist[v] = old_dist[v];
			new_clusters[v] = old_clusters[v];

			distance_t d;
			cu_for_star(he, mesh, v)
			{
				d = cu_update_step(mesh, old_dist, he);
				if(d < new_dist[v])
				{
					new_dist[v] = d;
					new_clusters[v] = old_dist[mesh->VT[cu_prev(he)]] < old_dist[mesh->VT[cu_next(he)]] ? old_clusters[mesh->VT[cu_prev(he)]] : old_clusters[mesh->VT[cu_next(he)]];
				}
			}
		}
	}
}

__device__
distance_t cu_update_step(CHE * mesh, const distance_t * dist, const index_t & he)
{
	index_t x[3];
	x[0] = mesh->VT[cu_next(he)];
	x[1] = mesh->VT[cu_prev(he)];
	x[2] = mesh->VT[he];

	vertex_cu X[2];
	X[0] = mesh->GT[x[0]] - mesh->GT[x[2]];
	X[1] = mesh->GT[x[1]] - mesh->GT[x[2]];

	distance_t t[2];
	t[0] = dist[x[0]];
	t[1] = dist[x[1]];

	distance_t q[2][2];
	q[0][0] = (X[0], X[0]);
	q[0][1] = (X[0], X[1]);
	q[1][0] = (X[1], X[0]);
	q[1][1] = (X[1], X[1]);
	
	distance_t det = q[0][0] * q[1][1] - q[0][1] * q[1][0];
	distance_t Q[2][2];
	Q[0][0] = q[1][1] / det;
	Q[0][1] = -q[0][1] / det;
	Q[1][0] = -q[1][0] / det;
	Q[1][1] = q[0][0] / det;

	distance_t delta = t[0] * (Q[0][0] + Q[1][0]) + t[1] * (Q[0][1] + Q[1][1]);
	distance_t dis = delta * delta - (Q[0][0] + Q[0][1] + Q[1][0] + Q[1][1]) * (t[0]*t[0]*Q[0][0] + t[0]*t[1]*(Q[1][0] + Q[0][1]) + t[1]*t[1]*Q[1][1] - 1);
	
	distance_t p;

	if(dis >= 0)
	{
		p = delta + sqrt(dis);
		p /= Q[0][0] + Q[0][1] + Q[1][0] + Q[1][1];
	}

	distance_t tp[2];
	tp[0] = t[0] - p;
	tp[1] = t[1] - p;

	vertex_cu n(tp[0] * (X[0][0]*Q[0][0] + X[1][0]*Q[1][0]) + tp[1] * (X[0][0]*Q[0][1] + X[1][0]*Q[1][1]),
			 tp[0] * (X[0][1]*Q[0][0] + X[1][1]*Q[1][0]) + tp[1] * (X[0][1]*Q[0][1] + X[1][1]*Q[1][1]),
			 tp[0] * (X[0][2]*Q[0][0] + X[1][2]*Q[1][0]) + tp[1] * (X[0][2]*Q[0][1] + X[1][2]*Q[1][1]) );

	distance_t cond[2];
	cond[0] = (X[0] , n);
	cond[1] = (X[1] , n);

	distance_t c[2];
	c[0] = cond[0] * Q[0][0] + cond[1] * Q[0][1];
	c[1] = cond[0] * Q[1][0] + cond[1] * Q[1][1];

	if(t[0] == INFINITY || t[1] == INFINITY || dis < 0 || c[0] >= 0 || c[1] >= 0)
	{
		distance_t dp[2];
		dp[0] = dist[x[0]] + *X[0];
		dp[1] = dist[x[1]] + *X[1];

		p = dp[dp[1] < dp[0]];
	}

	return p;
}

