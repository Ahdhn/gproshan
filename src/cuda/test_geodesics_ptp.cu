#include "hip/hip_runtime.h"
#include "test_geodesics_ptp.cuh"
#include "test_geodesics_ptp.h"

#include "geodesics_ptp.cuh"
#include "geodesics_ptp.h"

#include <fstream>
#include <hipblas.h>

distance_t * iter_error_parallel_toplesets_propagation_gpu(che * mesh, const vector<index_t> & sources, const vector<index_t> & limits, const index_t * sorted_index, const distance_t * exact_dist, float & time_ptp)
{
	hipDeviceReset();

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// BEGIN PTP

	CHE * h_mesh = new CHE(mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	distance_t * h_dist = new distance_t[h_mesh->n_vertices];

	distance_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(distance_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(distance_t) * h_mesh->n_vertices);

	index_t * d_sorted;
	hipMalloc(&d_sorted, sizeof(index_t) * h_mesh->n_vertices);

	distance_t * error = iter_error_run_ptp_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, sources, limits, sorted_index, d_sorted, exact_dist);
	
	delete [] h_dist;
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	hipFree(d_sorted);
	cuda_free_CHE(dd_mesh, d_mesh);

	// END PTP

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_ptp, start, stop);
	time_ptp /= 1000;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return error;
}

distance_t * iter_error_run_ptp_gpu(CHE * d_mesh, const index_t & n_vertices, distance_t * h_dist, distance_t ** d_dist, const vector<index_t> & sources, const vector<index_t> & limits, const index_t * h_sorted, index_t * d_sorted, const distance_t * exact_dist)
{
	#pragma omp parallel for
	for(index_t v = 0; v < n_vertices; v++)
		h_dist[v] = INFINITY;

	for(index_t i = 0; i < sources.size(); i++)
		h_dist[sources[i]] = 0;

	hipMemcpy(d_dist[0], h_dist, sizeof(distance_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_dist[1], h_dist, sizeof(distance_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_sorted, h_sorted, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);

	index_t d = 0, e = 0;
	index_t start, end;
	index_t iter = iterations(limits);

	distance_t * dist_error = new distance_t[iter - limits.size()];	

	for(index_t i = 2; i < iter; i++)
	{
		start = start_v(i, limits);
		end = end_v(i, limits);

		relax_ptp <<< NB(end - start), NT >>> (d_mesh, d_dist[!d], d_dist[d], d_sorted, end, start);
		hipMemcpy(h_dist, d_dist[!d], sizeof(distance_t) * n_vertices, hipMemcpyDeviceToHost);
		
		// calculating iteration error
		if(i >= limits.size())
		{
			distance_t & error = dist_error[e++] = 0;

			#pragma omp parallel for reduction(+: error)
			for(index_t v = 0; v < n_vertices; v++)
				if(exact_dist[v] > 0)
					error += abs(h_dist[v] - exact_dist[v]) / exact_dist[v];

			error /= n_vertices - sources.size();
		}

		d = !d;
	}

	return dist_error;
}


inline index_t farthest(distance_t * d, size_t n)
{
	index_t f = 0;

	#pragma omp parallel for
	for(index_t v = 0; v < n; v++)
		#pragma omp critical
		if(d[v] < INFINITY && d[f] < d[v])
			f = v;

	return f;
}

distance_t farthest_point_sampling_gpu(vector<index_t> & points, float & time, che * mesh, size_t n, distance_t radio)
{
	debug_me(GEODESICS_PTP)

	CHE * h_mesh;
	CHE * dd_mesh;
	CHE * d_mesh;

	h_mesh = new CHE(mesh);

	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	ofstream os((PATH_TEST + "fastmarching/" + mesh->name() + ".fps").c_str());

	size_t n_v = mesh->n_vertices();

	index_t * rings = new index_t[n_v];
	index_t * h_sorted = new index_t[n_v];

	distance_t * h_dist = new distance_t[n_v];

	distance_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(distance_t) * n_v);
	hipMalloc(&d_dist[1], sizeof(distance_t) * n_v);

	index_t * d_sorted;
	hipMalloc(&d_sorted, sizeof(index_t) * n_v);

	// ---------------------------------------------------------------------------------------------
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	if(n >= mesh->n_vertices())
		n = mesh->n_vertices() / 2;

	n -= points.size();
	points.reserve(n);

	time = 0;
	float time_aux;

	index_t d;
	int f;
	distance_t max_dis = INFINITY;
	while(n-- && max_dis > radio)
	{
		hipEventRecord(start, 0);

		vector<index_t> limites;
		mesh->compute_toplesets(rings, h_sorted, limites, points);

		d = run_ptp_gpu(d_mesh, n_v, h_dist, d_dist, points, limites, h_sorted, d_sorted);

		// 1 indexing T_T
		#ifdef SINGLE_P
		hipblasIsamax(handle, mesh->n_vertices(), d_dist[d], 1, &f);
		#else
		hipblasIdamax(handle, mesh->n_vertices(), d_dist[d], 1, &f);
		#endif

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_aux, start, stop);

		time_aux /= 1000;
		time += time_aux;

		os << points.size() << " " << time_aux << endl;

		if(radio > 0 || !n)
			hipMemcpy(&max_dis, d_dist[d] + f - 1, sizeof(distance_t), hipMemcpyDeviceToHost);
		points.push_back(f - 1);
	}


	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipblasDestroy(handle);
	// ---------------------------------------------------------------------------------------------

	os.close();

	delete [] rings;
	delete [] h_sorted;
	delete [] h_dist;

	cuda_free_CHE(dd_mesh, d_mesh);
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	hipFree(d_sorted);

	return max_dis;
}

