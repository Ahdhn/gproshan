#include "test_geodesics_ptp.cuh"
#include "test_geodesics_ptp.h"

#include "geodesics_ptp.cuh"
#include "geodesics_ptp.h"

#include <fstream>
#include <hipblas.h>

distance_t * iter_error_parallel_toplesets_propagation_gpu(che * mesh, const vector<index_t> & sources, const vector<index_t> & limits, const index_t * sorted_index, const distance_t * exact_dist, double & time_ptp)
{
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// BEGIN PTP

	CHE * h_mesh = new CHE(mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	distance_t * h_dist = new distance_t[h_mesh->n_vertices];

	distance_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(distance_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(distance_t) * h_mesh->n_vertices);

	index_t * d_sorted;
	hipMalloc(&d_sorted, sizeof(index_t) * h_mesh->n_vertices);

	distance_t * error = iter_error_run_ptp_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, sources, limits, sorted_index, d_sorted, exact_dist);
	
	delete [] h_dist;
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	hipFree(d_sorted);
	cuda_free_CHE(dd_mesh, d_mesh);

	// END PTP

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	time_ptp = time / 1000;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	return error;
}

/// Return an array of time in seconds.
double * times_farthest_point_sampling_ptp_gpu(che * mesh, vector<index_t> & samples, size_t n, distance_t radio)
{
	hipDeviceReset();

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// BEGIN FPS PTP

	CHE * h_mesh = new CHE(mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	distance_t * h_dist = new distance_t[h_mesh->n_vertices];

	distance_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(distance_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(distance_t) * h_mesh->n_vertices);

	distance_t * d_error;
	hipMalloc(&d_error, sizeof(distance_t) * h_mesh->n_vertices);

	index_t * d_sorted;
	hipMalloc(&d_sorted, sizeof(index_t) * h_mesh->n_vertices);

	vector<index_t> limits;
	index_t * toplesets = new index_t[h_mesh->n_vertices];
	index_t * sorted_index = new index_t[h_mesh->n_vertices];

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	if(n >= h_mesh->n_vertices) n = h_mesh->n_vertices >> 1;

	double * times = new double[n + 1];

	n -= samples.size();
	samples.reserve(n);

	float time_fps;
	index_t d;
	int f;
	distance_t max_dist = INFINITY;
	while(n-- && max_dist > radio)
	{
		hipEventRecord(start, 0);
		
		limits.clear();
		mesh->compute_toplesets(toplesets, sorted_index, limits, samples);

		d = run_ptp_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, samples, limits, sorted_index, d_sorted, d_error);

		// 1 indexing
		#ifdef SINGLE_P
			hipblasIsamax(handle, mesh->n_vertices(), d_dist[d], 1, &f);
		#else
			hipblasIdamax(handle, mesh->n_vertices(), d_dist[d], 1, &f);
		#endif
		
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_fps, start, stop);

		times[samples.size()] = time_fps / 1000;

		if(radio > 0 || !n)
			hipMemcpy(&max_dist, d_dist[d] + f - 1, sizeof(distance_t), hipMemcpyDeviceToHost);

		samples.push_back(f - 1);
	}

	hipblasDestroy(handle);

	delete [] h_dist;
	delete [] toplesets;
	delete [] sorted_index;
	
	hipFree(d_error);
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	hipFree(d_sorted);
	cuda_free_CHE(dd_mesh, d_mesh);

	// END FPS PTP

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return times;
}

distance_t * iter_error_run_ptp_gpu(CHE * d_mesh, const index_t & n_vertices, distance_t * h_dist, distance_t ** d_dist, const vector<index_t> & sources, const vector<index_t> & limits, const index_t * h_sorted, index_t * d_sorted, const distance_t * exact_dist)
{
	#pragma omp parallel for
	for(index_t v = 0; v < n_vertices; v++)
		h_dist[v] = INFINITY;

	for(index_t i = 0; i < sources.size(); i++)
		h_dist[sources[i]] = 0;

	hipMemcpy(d_dist[0], h_dist, sizeof(distance_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_dist[1], h_dist, sizeof(distance_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_sorted, h_sorted, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);

	index_t d = 0, e = 0;
	index_t start, end;
	index_t iter = iterations(limits);

	distance_t * dist_error = new distance_t[iter - limits.size()];	

	for(index_t i = 2; i < iter; i++)
	{
		start = start_v(i, limits);
		end = end_v(i, limits);

		if(end == start) break;

		relax_ptp <<< NB(end - start), NT >>> (d_mesh, d_dist[!d], d_dist[d], d_sorted, end, start);
		hipMemcpy(h_dist, d_dist[!d], sizeof(distance_t) * n_vertices, hipMemcpyDeviceToHost);
		
		// calculating iteration error
		if(i >= limits.size())
			dist_error[e++] = compute_error(h_dist, exact_dist, n_vertices, sources.size());

		d = !d;
	}

	return dist_error;
}

