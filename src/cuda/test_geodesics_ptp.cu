#include "hip/hip_runtime.h"
#include "test_geodesics_ptp.cuh"
#include "test_geodesics_ptp.h"

#include "geodesics_ptp.cuh"
#include "geodesics_ptp.h"

#include <fstream>
#include <omp.h>
#include <hipblas.h>

distance_t * iter_error_parallel_toplesets_propagation_gpu(che * mesh, const vector<index_t> & sources, const vector<index_t> & limits, const index_t * sorted_index, const distance_t * exact_dist, double & time_ptp)
{
	hipDeviceReset();
/*
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
*/
	TIC(time_ptp)

	// BEGIN PTP

	CHE * h_mesh = new CHE(mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	distance_t * h_dist = new distance_t[h_mesh->n_vertices];

	distance_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(distance_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(distance_t) * h_mesh->n_vertices);

	index_t * d_sorted;
	hipMalloc(&d_sorted, sizeof(index_t) * h_mesh->n_vertices);

	distance_t * error = iter_error_run_ptp_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, sources, limits, sorted_index, d_sorted, exact_dist);
	
	delete [] h_dist;
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	hipFree(d_sorted);
	cuda_free_CHE(dd_mesh, d_mesh);

	// END PTP

	TOC(time_ptp)
/*
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_ptp, start, stop);
	time_ptp /= 1000;

	hipEventDestroy(start);
	hipEventDestroy(stop);
*/
	return error;
}

/// Return an array of time in seconds.
double * times_farthest_point_sampling_ptp_gpu(che * mesh, vector<index_t> & samples, size_t n, distance_t radio)
{
	hipDeviceReset();

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// BEGIN FPS PTP

	CHE * h_mesh = new CHE(mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	distance_t * h_dist = new distance_t[h_mesh->n_vertices];

	distance_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(distance_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(distance_t) * h_mesh->n_vertices);

	index_t * d_sorted;
	hipMalloc(&d_sorted, sizeof(index_t) * h_mesh->n_vertices);

	vector<index_t> limits;
	index_t * toplesets = new index_t[h_mesh->n_vertices];
	index_t * sorted_index = new index_t[h_mesh->n_vertices];

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	if(n >= h_mesh->n_vertices) n = h_mesh->n_vertices >> 1;

	double * times = new double[n + 1];

	n -= samples.size();
	samples.reserve(n);

	float time_fps;
	index_t d;
	int f;
	distance_t max_dist = INFINITY;
	while(n-- && max_dist > radio)
	{
		hipEventRecord(start, 0);
		
		limits.clear();
		mesh->compute_toplesets(toplesets, sorted_index, limits, samples);

		d = run_ptp_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, samples, limits, sorted_index, d_sorted);

		// 1 indexing
		#ifdef SINGLE_P
			hipblasIsamax(handle, mesh->n_vertices(), d_dist[d], 1, &f);
		#else
			hipblasIdamax(handle, mesh->n_vertices(), d_dist[d], 1, &f);
		#endif
		
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_fps, start, stop);

		times[samples.size()] = time_fps / 1000;

		if(radio > 0 || !n)
			hipMemcpy(&max_dist, d_dist[d] + f - 1, sizeof(distance_t), hipMemcpyDeviceToHost);

		samples.push_back(f - 1);
	}

	hipblasDestroy(handle);

	delete [] h_dist;
	delete [] toplesets;
	delete [] sorted_index;

	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	hipFree(d_sorted);
	cuda_free_CHE(dd_mesh, d_mesh);

	// END FPS PTP

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return times;
}

distance_t * iter_error_run_ptp_gpu(CHE * d_mesh, const index_t & n_vertices, distance_t * h_dist, distance_t ** d_dist, const vector<index_t> & sources, const vector<index_t> & limits, const index_t * h_sorted, index_t * d_sorted, const distance_t * exact_dist)
{
	#pragma omp parallel for
	for(index_t v = 0; v < n_vertices; v++)
		h_dist[v] = INFINITY;

	for(index_t i = 0; i < sources.size(); i++)
		h_dist[sources[i]] = 0;

	hipMemcpy(d_dist[0], h_dist, sizeof(distance_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_dist[1], h_dist, sizeof(distance_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_sorted, h_sorted, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);

	index_t d = 0, e = 0;
	index_t start, end;
	index_t iter = iterations(limits);

	distance_t * dist_error = new distance_t[iter - limits.size()];	

	for(index_t i = 2; i < iter; i++)
	{
		start = start_v(i, limits);
		end = end_v(i, limits);

		relax_ptp <<< NB(end - start), NT >>> (d_mesh, d_dist[!d], d_dist[d], d_sorted, end, start);
		hipMemcpy(h_dist, d_dist[!d], sizeof(distance_t) * n_vertices, hipMemcpyDeviceToHost);
		
		// calculating iteration error
		if(i >= limits.size())
		{
			distance_t & error = dist_error[e++] = 0;

			#pragma omp parallel for reduction(+: error)
			for(index_t v = 0; v < n_vertices; v++)
				if(exact_dist[v] > 0)
					error += abs(h_dist[v] - exact_dist[v]) / exact_dist[v];

			error /= n_vertices - sources.size();
		}

		d = !d;
	}

	return dist_error;
}

